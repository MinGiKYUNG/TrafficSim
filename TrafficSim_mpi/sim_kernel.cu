
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>

using namespace std;

#define THREADS_PER_BLOCK 512


typedef struct _MyData{
	char name[8];
        int my_num;
}MyData;

__global__ void dot(MyData* my_data, int rank, int data_size) {
	for(int i = 0 ; i < data_size ; i++){
		my_data[i].my_num *= rank;
		printf("%dprocess[%d]:%d \n", rank, i, my_data[i].my_num);
	}
}

void runGPU(vector<MyData>* v, int rank){
	MyData* dev_data;
	MyData* host_data;

	host_data = &(v->at(0));
        
	hipMalloc((void**) & dev_data, v->size() * sizeof(MyData));

	hipMemcpy(dev_data, host_data, v->size() * sizeof(MyData), hipMemcpyHostToDevice);

	for (int i = 0; i < v->size(); i++) {
		//printf("%dprocess[%d]:%d \n", rank, i, host_data[i].my_num);
	}

	dot<<< 1, 1 >>>(dev_data, rank, v->size());


	hipMemcpy(host_data, dev_data, v->size() * sizeof(MyData), hipMemcpyDeviceToHost);


	FILE *fp = fopen("cuda_mpi.txt", "w");
	for (int i = 0; i < v->size(); i++) {
		fprintf(fp, "%dprocess[%d]:%d \n", rank, i, host_data[i].my_num);
	}
	fclose(fp);	

	hipFree(dev_data);
}
